#include "hip/hip_runtime.h"
#include "device.cuh"

__global__ void Device::set_dataset(float *device_input, float *data_set,
                                    unsigned int input_size) {
  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid_x < input_size) {

    device_input[tid_x] = data_set[tid_x];
  }
}

__global__ void Device::set_layer_memory(float *device_delta,
                                         float *device_prvdeltas,
                                         unsigned int input_size,
                                         unsigned int neuron_size) {

  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid_x < neuron_size) {

    device_delta[tid_x] = 0.0f;

    unsigned int index = tid_x * input_size;

    for (unsigned int i = 0; i < input_size; i++) {
      device_prvdeltas[i + index] = 0.0f;
    }
  }
}

__global__ void
Device::neuron_update_layer(float *device_input, float *device_weights,
                          float *device_delta, float *device_prvdeltas,
                          float learning_rate, float momentum,
                          unsigned int input_size, unsigned int neuron_size) {

  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid_x < neuron_size * input_size) {

    float dw;

    dw = learning_rate * device_input[tid_x % input_size] *
         device_delta[(int)floorf((float)tid_x / (float)input_size)];

    dw += momentum * device_prvdeltas[tid_x];

    device_prvdeltas[tid_x] = dw;
    device_weights[tid_x] += (1-momentum)*dw;
  }
}
__global__ void Device::neuron_propagate_inlayer(
    float *device_input, float *nl_device_input, float *device_weights,
    float *device_wbias, unsigned int input_size, unsigned int neuron_size) {

  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid_x < neuron_size) {

    float output = 0;
    unsigned int index = tid_x * input_size;

    for (unsigned int j = 0; j < input_size; j++) {
      output += device_weights[j + index] * device_input[j];
    }

    output += device_wbias[tid_x];

    float res = 1 / (1 + expf(-output));
    nl_device_input[tid_x] = res;
  }
}

__global__ void Device::neuron_propagate_layer(
    float *device_input, float *nl_device_input, float *device_weights,
    float *device_wbias, unsigned int input_size, unsigned int neuron_size) {

  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid_x < neuron_size) {

    float output = 0;
    unsigned int index = tid_x * input_size;

    for (unsigned int j = 0; j < input_size; j++) {

      output += device_weights[j + index] * device_input[j];
    }

    output += device_wbias[tid_x];

    float res = 1 / (1 + expf(-output));
    nl_device_input[tid_x] = res;
  }
}

__global__ void
Device::neuron_outlayer_train(float *device_output, float *device_delta,
                            float *device_wbias, float *device_awaited_output,
                            float learning_rate, unsigned int input_size,
                            unsigned int neuron_size) {

  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid_x < neuron_size) {

    float out = device_output[tid_x];

    float delta = (device_awaited_output[tid_x] - out) * out * (1 - out);
    device_delta[tid_x] = delta;
    device_wbias[tid_x] += learning_rate * delta;
  }
}

__inline__ __device__ float warpReduceSum(float val) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2)
    val += __shfl_down(val, offset);
  return val;
}

__inline__ __device__ float blockReduceSum(float val) {

  static __shared__ float shared[32];
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);

  if (lane == 0)
    shared[wid] = val;

  __syncthreads();

  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

  if (wid == 0)
    val = warpReduceSum(val);

  return val;
}

__global__ void Device::reduction(float *data, float *out_data,
                                  unsigned int size) {
  float sum = 0;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
       i += blockDim.x * gridDim.x) {
    sum += data[i];
  }
  sum = blockReduceSum(sum);
  if (threadIdx.x == 0)
    out_data[blockIdx.x] = sum;
}

__global__ void Device::neuron_layer_delta(float *device_delta_summands,
                                         float *pl_device_weights,
                                         float *pl_device_delta,
                                         unsigned int input_size,
                                         unsigned int neuron_size) {
  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid_x < neuron_size * input_size) {
    device_delta_summands[tid_x] =
        pl_device_weights[tid_x] *
        pl_device_delta[(int)floorf((float)tid_x / (float)input_size)];
  }
}

__global__ void Device::neuron_layer_train(
    float *device_output, float *device_delta_summands, float *device_wbias,
    float *device_delta, float *device_awaited_output, float learning_rate,
    unsigned int pl_input_size, unsigned int pl_neuron_size,
    unsigned int input_size, unsigned int neuron_size) {

  unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid_x < neuron_size) {

    float out;
    /*float delta = 0;

    for (unsigned int i = 0; i < pl_neuron_size; i++) {
      for (unsigned int j = 0; j < pl_input_size; j++) {
        delta += pl_device_weights[i * pl_input_size + j] * pl_device_delta[i];
      }
    }*/
    float delta = device_delta_summands[0];

    out = device_output[tid_x];

    float rdelta = out * (1 - out) * delta;
    device_delta[tid_x] = rdelta;
    device_wbias[tid_x] += learning_rate * rdelta;
  }
}
