#include "hip/hip_runtime.h"
#include "device.cuh"

__global__ void
Device::tile_update_layer(float *device_input, float *device_weights,
                          float *device_delta, float *device_prvdeltas,
                          float learning_rate, float momentum,
                          unsigned int input_offset, unsigned int neuron_offset,
                          unsigned int input_size, unsigned int weight_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;

  float dw;
  for (unsigned int j = 0; j < input_size; j++) {

    unsigned int index = j + weight_offset + tid_x * input_size;

    dw = learning_rate * device_input[j + input_offset] *
         device_delta[neuron_offset + tid_x];
    dw += momentum * device_prvdeltas[index];

    device_prvdeltas[index] = dw;
    device_weights[index] += dw;
  }
}
__global__ void Device::tile_propagate_inlayer(
    float *data_set, float *device_input, float *device_weights,
    float *device_wbias, unsigned int input_size, unsigned int neuron_size) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;

  float output = 0;
  for (unsigned int j = 0; j < input_size; j++) {
    device_input[j] = data_set[j];
    output += device_weights[j + tid_x * input_size] * device_input[j];
  }

  output += device_wbias[tid_x];

  float res = 1 / (1 + exp(-output));
  device_input[input_size + tid_x] = res;
}

__global__ void Device::tile_propagate_layer(
    float *device_input, float *device_weights, float *device_wbias,
    unsigned int input_size, unsigned int input_offset,
    unsigned int neuron_offset, unsigned int output_offset,
    unsigned int weight_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned int index = neuron_offset + tid_x;

  float output = 0;
  for (unsigned int j = 0; j < input_size; j++) {

    output += device_weights[j + weight_offset + tid_x * input_size] *
              device_input[j + input_offset];
  }

  output += device_wbias[input_size + tid_x];

  float res = 1 / (1 + exp(-output));
  device_input[output_offset + tid_x] = res;
}

__global__ void
Device::tile_outlayer_train(float *device_input, float *device_delta,
                            float *device_wbias, float *device_awaited_output,
                            float learning_rate, unsigned int output_offset,
                            unsigned int nl_neuron_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned int index = nl_neuron_offset + tid_x;
  float out;

  out = device_input[output_offset + tid_x];
  float delta = (device_awaited_output[tid_x] - out) * out * (1 - out);
  device_delta[index] = delta;
  device_wbias[index] += learning_rate * delta;
}

__global__ void Device::tile_layer_train(
    float *device_input, float *device_weights, float *device_wbias,
    float *device_delta, float *device_awaited_output, float learning_rate,
    unsigned int pl_neuron_size, unsigned int pl_input_size,
    unsigned int tl_weight_offset, unsigned int tl_neuron_offset,
    unsigned int tl_input_size, unsigned int nl_neuron_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned int index = nl_neuron_offset + tid_x;

  float out;
  float delta = 0;

  for (unsigned int i = 0; i < pl_neuron_size; i++) {
    for (unsigned int j = 0; j < pl_input_size; j++) {
      delta += device_weights[tl_weight_offset + i * pl_input_size + j] *
               device_delta[tl_neuron_offset + i];
    }
  }

  out = device_input[index + tl_input_size];

  float rdelta = out * (1 - out) * delta;
  device_delta[index] = rdelta;
  device_wbias[index] += learning_rate * rdelta;
}
