#include "hip/hip_runtime.h"
#include "gpunetwork.cuh"

__global__ void
Device::tile_update_layer(float *device_input, float *device_weights,
                          unsigned int input_offset, unsigned int neuron_offset,
                          float learning_rate, float momentum,
                          float *device_delta, float *device_prvdeltas,
                          unsigned int input_size, unsigned int weight_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;

  float dw;
  for (unsigned int j = 0; j < input_size; j++) {
    dw = learning_rate * device_input[j + input_offset] *
         device_delta[neuron_offset + tid_x];
    dw += momentum * device_prvdeltas[j + weight_offset + tid_x * input_size];

    device_prvdeltas[j + weight_offset + tid_x * input_size] = dw;
    device_weights[j + weight_offset + tid_x * input_size] += dw;
  }
}

__global__ void Device::tile_propagate_layer(
    float *device_input, float *device_weights, float *device_wbias,
    float *device_output, unsigned int input_size, unsigned int neuron_size,
    unsigned int input_offset, unsigned int neuron_offset,
    unsigned int weight_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;

  float output = 0;
  for (unsigned int j = 0; j < input_size; j++) {
    output += device_weights[j + weight_offset + tid_x * input_size] *
              device_input[j + input_offset];
  }

  output += device_wbias[tid_x + neuron_offset];
  /*printf("threadIdx: %d input_offset: %d neuron_offset: %d\n", tid_x,
         input_offset, neuron_offset);*/
  float res = 1 / (1 + exp(-output));
  device_output[neuron_offset + tid_x] = res;
  device_input[neuron_offset + tid_x] = res;
}

__global__ void Device::tile_outlayer_train(
    float *device_wbias, float *device_output, float *device_awaited_output,
    unsigned int neuron_offset, float learning_rate, float *device_delta) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;
  float out;
  out = device_output[tid_x + neuron_offset];

  device_delta[tid_x + neuron_offset] =
      (device_awaited_output[tid_x] - out) * out * (1 - out);

  device_wbias[tid_x + neuron_offset] +=
      learning_rate * (device_awaited_output[tid_x] - out) * out * (1 - out);
}

__global__ void
Device::tile_layer_train(float *device_weights, float *device_wbias,
                         float *device_output, float *device_awaited_output,
                         unsigned int neuron_offset, float learning_rate,
                         float *device_delta, unsigned int layer_offset,
                         unsigned int input_size, unsigned int weight_offset,
                         unsigned int weight_layer_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;

  float out;
  float delta = 0;

  for (unsigned int j = 0; j < input_size; j++) {
    delta += device_weights[j + weight_layer_offset + tid_x * input_size] *
             device_delta[layer_offset + tid_x];
  }

  out = device_output[tid_x + neuron_offset];
  /*printf("threadIdx: %d neuron_offset: %d layer_offset: %d\n", tid_x,
         neuron_offset, layer_offset);*/
  device_delta[tid_x + neuron_offset] = out * (1 - out) * delta;
  device_wbias[tid_x + neuron_offset] +=
      learning_rate * out * (1 - out) * delta;
}

void GPUNetwork::init_network(unsigned int *inputs, unsigned int *neurons,
                              unsigned int clayers) {

  count_layers = clayers;
  arr_input_size = new unsigned int[clayers];
  arr_neuron_size = new unsigned int[clayers];
  sum_input_size = new unsigned int[clayers + 1];
  sum_neuron_size = new unsigned int[clayers + 1];
  sum_weight_size = new unsigned int[clayers + 1];
  sum_input_size[0] = 0;
  sum_neuron_size[0] = 0;
  sum_weight_size[0] = 0;
  num_blocks = new int[clayers];
  threads_per_block = new int[clayers];

  std::memcpy(arr_input_size, inputs, sizeof(unsigned int) * clayers);
  std::memcpy(arr_neuron_size, neurons, sizeof(unsigned int) * clayers);

  hipDeviceProp_t device_props;
  checkErrorsCuda(hipGetDeviceProperties(&device_props, 0));

  size_t input_size = 0;
  size_t neuron_size = 0;
  size_t weight_size = 0;

  for (int l = 0; l < clayers; l++) {
    input_size += inputs[l];
    neuron_size += neurons[l];

    weight_size += inputs[l] * neurons[l];
    if (neurons[l] > device_props.maxThreadsPerBlock)
      threads_per_block[l] = device_props.maxThreadsPerBlock;
    else
      threads_per_block[l] = neurons[l];

    num_blocks[l] = (int)(neurons[l] / threads_per_block[l]);

    if (num_blocks[l] == 0)
      num_blocks[l]++;
    else if (0 != neurons[l] % num_blocks[l])
      num_blocks[l]++;

    std::cout << "num_blocks = " << num_blocks[l] << std::endl;
  }

  for (int l = 1; l < clayers + 1; l++) {
    sum_input_size[l] = inputs[l - 1] + sum_input_size[l - 1];
    sum_neuron_size[l] = neurons[l - 1] + sum_neuron_size[l - 1];
    sum_weight_size[l] = neurons[l - 1] * inputs[l - 1];
  }

  for (int l = 0; l < clayers + 1; l++) {
    std::cout << l << " | " << sum_input_size[l] << " | " << sum_neuron_size[l]
              << std::endl;
  }

  std::cout << input_size << " | " << neuron_size << " | " << weight_size
            << std::endl
            << "hipMalloc" << std::endl;

  checkErrorsCuda(hipMalloc(&device_input, sizeof(float) * input_size));

  checkErrorsCuda(hipMalloc(&device_wbias, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMalloc(&device_output, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMalloc(&device_delta, sizeof(float) * neuron_size));

  checkErrorsCuda(hipMalloc(&device_weights, sizeof(float) * weight_size));
  checkErrorsCuda(hipMalloc(&device_prvdeltas, sizeof(float) * weight_size));

  std::cout << "hipMemset" << std::endl;

  checkErrorsCuda(hipMemset(device_input, 0, sizeof(float) * input_size));

  checkErrorsCuda(hipMemset(device_wbias, 0, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMemset(device_output, 0, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMemset(device_delta, 0, sizeof(float) * neuron_size));

  checkErrorsCuda(hipMemset(device_weights, 0, sizeof(float) * weight_size));
  checkErrorsCuda(hipMemset(device_prvdeltas, 0, sizeof(float) * weight_size));
}

void GPUNetwork::propagate_network(const float *input) {

  checkErrorsCuda(hipMemcpy(device_input, input, arr_input_size[0] * sizeof(float),
                             hipMemcpyHostToDevice));

  for (unsigned int l = 0; l < count_layers; l++) {
    Device::tile_propagate_layer<<<num_blocks[l], threads_per_block[l]>>>(
        device_input, device_weights, device_wbias, device_output,
        arr_input_size[l], arr_neuron_size[l], sum_input_size[l], sum_neuron_size[l],
        sum_weight_size[l]);
    hipDeviceSynchronize();
  }
}

float GPUNetwork::train_network(const float *input, const float *awaited_output,
                                const float learning_rate, float momentum) {

  propagate_network(input);
  float *device_awaited_output;
  checkErrorsCuda(hipMalloc(&device_awaited_output,
                             sizeof(float) * arr_neuron_size[count_layers - 1]));
  checkErrorsCuda(hipMemcpy(device_awaited_output, awaited_output,
                             sizeof(float) * arr_neuron_size[count_layers - 1],
                             hipMemcpyHostToDevice));

  float total_error = 0;
  float *out;
  out = getOutput();

  for (unsigned int i = 0; i < arr_neuron_size[count_layers - 1]; i++) {
    total_error +=
        0.5 * (awaited_output[i] - out[i]) * (awaited_output[i] - out[i]);
  }

  Device::tile_outlayer_train<<<num_blocks[count_layers - 1],
                                threads_per_block[count_layers - 1]>>>(
      device_wbias, device_output, device_awaited_output,
      sum_neuron_size[count_layers - 1], momentum, device_delta);

  for (int l = count_layers - 2; l >= 0; l--) {
    Device::tile_layer_train<<<num_blocks[l], threads_per_block[l]>>>(
        device_weights, device_wbias, device_output, device_awaited_output,
        sum_neuron_size[l], learning_rate, device_delta, sum_neuron_size[l + 1],
        arr_input_size[l+1], sum_weight_size[l], sum_weight_size[l + 1]);
  }

  for (unsigned int l = 0; l < count_layers; l++) {
    Device::tile_update_layer<<<num_blocks[l], threads_per_block[l]>>>(
        device_input, device_weights, sum_input_size[l], sum_neuron_size[l],
        learning_rate, learning_rate, device_delta, device_prvdeltas,
        arr_input_size[l], sum_weight_size[l]);
  }

  delete out;
  // checkErrorsCuda(hipFree(device_awaited_output));

  return total_error;
}

float *GPUNetwork::getOutput() {
  float *out = new float[arr_neuron_size[count_layers - 1]];

  checkErrorsCuda(hipMemcpy(
      out, device_output + sum_neuron_size[count_layers - 1],
      arr_neuron_size[count_layers - 1] * sizeof(float), hipMemcpyDeviceToHost));
  return out;
}

GPUNetwork::~GPUNetwork() {
  delete num_blocks;
  delete threads_per_block;
  delete arr_input_size;
  delete arr_neuron_size;
  delete sum_input_size;
  delete sum_neuron_size;
  delete sum_weight_size;

  // device_land
  checkErrorsCuda(hipFree(device_input));

  checkErrorsCuda(hipFree(device_weights));
  checkErrorsCuda(hipFree(device_wbias));

  checkErrorsCuda(hipFree(device_delta));
  checkErrorsCuda(hipFree(device_prvdeltas));
  checkErrorsCuda(hipFree(device_output));
}
