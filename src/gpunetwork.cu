#include "hip/hip_runtime.h"
#include "gpunetwork.cuh"

void GPUNetwork::init_network(unsigned int *inputs, unsigned int *neurons,
                              unsigned int clayers) {

  count_layers = clayers;
  num_blocks = new int[clayers];
  threads_per_block = new int[clayers];

  mul_num_blocks = new int[clayers];
  mul_threads_per_block = new int[clayers];

  neuron_size = new int[clayers];
  input_size = new int[clayers];

  std::memcpy(neuron_size, neurons, sizeof(int) * clayers);
  std::memcpy(input_size, inputs, sizeof(int) * clayers);

  for (unsigned int i = 0; i < clayers; i++) {
    std::cout << "Neuron_Size " << i << " " << neuron_size[i] << std::endl;
    std::cout << "Input_Size " << i << " " << input_size[i] << std::endl;
  }

  hipDeviceProp_t device_props;
  checkErrorsCuda(hipGetDeviceProperties(&device_props, 0));
  for (unsigned int l = 0; l < clayers; l++) {
    if (neurons[l] > (unsigned int)device_props.maxThreadsPerBlock)
      threads_per_block[l] = device_props.maxThreadsPerBlock;
    else
      threads_per_block[l] = neurons[l];

    num_blocks[l] = (int)(neurons[l] / threads_per_block[l]);

    if (num_blocks[l] == 0) {
      num_blocks[l]++;
    } else if (0 != neurons[l] % num_blocks[l]) {
      num_blocks[l]++;
    }
    if (neurons[l] * inputs[l] > (unsigned int)device_props.maxThreadsPerBlock)
      mul_threads_per_block[l] = device_props.maxThreadsPerBlock;
    else
      mul_threads_per_block[l] = neurons[l] * inputs[l];

    mul_num_blocks[l] =
        (int)(neurons[l] * inputs[l] / mul_threads_per_block[l]);

    if (mul_num_blocks[l] == 0) {
      mul_num_blocks[l]++;
    } else if (0 != neurons[l] * inputs[l] % mul_num_blocks[l]) {
      mul_num_blocks[l]++;
    }

    // std::cout << "num_blocks = " << num_blocks[l] << std::endl;
  }
  device_inputs = new float *[clayers];

  device_weights = new float *[clayers];
  device_wbias = new float *[clayers];

  device_delta = new float *[clayers];
  device_prvdeltas = new float *[clayers];

  device_delta_summands = new float *[clayers];
  device_delta_summands_out = new float *[clayers];

  checkErrorsCuda(hipMalloc((void **)&device_output,
                             sizeof(float) * neurons[count_layers - 1]));

  for (unsigned int i = 0; i < clayers; i++) {
    checkErrorsCuda(
        hipMalloc((void **)&device_inputs[i], sizeof(float) * inputs[i]));
    checkErrorsCuda(
        hipMalloc((void **)&device_wbias[i], sizeof(float) * neurons[i]));
    checkErrorsCuda(
        hipMalloc((void **)&device_delta[i], sizeof(float) * neurons[i]));

    checkErrorsCuda(hipMalloc((void **)&device_weights[i],
                               sizeof(float) * inputs[i] * neurons[i]));
    checkErrorsCuda(hipMalloc((void **)&device_prvdeltas[i],
                               sizeof(float) * inputs[i] * neurons[i]));
    checkErrorsCuda(hipMalloc((void **)&device_delta_summands[i],
                               sizeof(float) * inputs[i] * neurons[i]));
    checkErrorsCuda(hipMalloc((void **)&device_delta_summands_out[i],
                               sizeof(float) * inputs[i] * neurons[i]));

    Device::set_layer_memory<<<num_blocks[i], threads_per_block[i]>>>(
        device_delta[i], device_prvdeltas[i], inputs[i], neurons[i]);

    float *weights = new float[inputs[i] * neurons[i]];
    float *wbias = new float[neurons[i]];

    for (unsigned int j = 0; j < inputs[i] * neurons[i]; j++) {
      weights[j] = -0.5 +
                    static_cast<float>(rand()) /
                        (static_cast<float>(RAND_MAX / (0.5 - (-0.5))));
    }
    for (unsigned int j = 0; j < neurons[i]; j++) {
      wbias[j] = -0.5 +
                 static_cast<float>(rand()) /
                     (static_cast<float>(RAND_MAX / (0.5 - (-0.5))));
    }

    checkErrorsCuda(hipMemcpy(device_weights[i], weights,
                               sizeof(float) * inputs[i] * neurons[i],
                               hipMemcpyHostToDevice));
    checkErrorsCuda(hipMemcpy(device_wbias[i], wbias,
                               sizeof(float) * neurons[i],
                               hipMemcpyHostToDevice));
    delete weights;
    delete wbias;
  }
}

unsigned int GPUNetwork::propagate_network(float *data_set, float *label_set,
                                           unsigned int dataset_count,
                                           size_t set_size, size_t label_size) {
  unsigned int success = 0;
  unsigned int success0 = 0;
  unsigned int success1 = 0;
  unsigned int success2 = 0;
  unsigned int success3 = 0;
  unsigned int success4 = 0;
  unsigned int success5 = 0;
  unsigned int success6 = 0;
  unsigned int success7 = 0;
  unsigned int success8 = 0;
  unsigned int success9 = 0;

  checkErrorsCuda(
      hipMalloc((void **)&test_device_dataset, sizeof(float) * set_size));

  checkErrorsCuda(hipMemcpy(test_device_dataset, data_set,
                             sizeof(float) * set_size, hipMemcpyHostToDevice));

  for (unsigned int i = 0; i < dataset_count; i++) {

    Device::set_dataset<<<1, (set_size / dataset_count)>>>(
        device_inputs[0],
        test_device_dataset + (i * (set_size / dataset_count)), input_size[0]);
    // checkErrorsCuda(hipDeviceSynchronize());

    Device::tile_propagate_inlayer<<<num_blocks[0], threads_per_block[0]>>>(
        device_inputs[0], device_inputs[1], device_weights[0], device_wbias[0],
        input_size[0], neuron_size[0]);
    // checkErrorsCuda(hipDeviceSynchronize());

    for (unsigned int l = 1; l < count_layers; l++) {
      if (l >= count_layers - 1) {
        // std::cout << "l115 " << l << std::endl;
        Device::tile_propagate_layer<<<num_blocks[l], threads_per_block[l]>>>(
            device_inputs[l], device_output, device_weights[l], device_wbias[l],
            input_size[l], neuron_size[l]);
      } else {
        // std::cout << "l121 " << l << std::endl;
        Device::tile_propagate_layer<<<num_blocks[l], threads_per_block[l]>>>(
            device_inputs[l], device_inputs[l + 1], device_weights[l],
            device_wbias[l], input_size[l], neuron_size[l]);
      }
      // checkErrorsCuda(hipDeviceSynchronize());
    }

    float *out;
    out = getOutput();

    float outf = -1;
    float index = 0;
    float desired = 0;

    for (int j = 0; j < 10; j++) {
      if (out[j] > outf) {
        outf = out[j];
        index = j;
      }
    }

    for (int j = 0; j < 10; j++) {
      if (label_set[j + i * (label_size / dataset_count)] == 1) {
        desired = j;
      }
    } /*
 for (int j = 0; j < 10; j++) {
   index += out[j];
   desired += label_set[j + i * (label_size / dataset_count)];
 }*/

    if ((int)std::round(index) == desired) {
      success++;
      if (desired == 0) {
        success0++;
      } else if (desired == 1) {
        success1++;
      } else if (desired == 2) {
        success2++;
      } else if (desired == 3) {
        success3++;
      } else if (desired == 4) {
        success4++;
      } else if (desired == 5) {
        success5++;
      } else if (desired == 6) {
        success6++;
      } else if (desired == 7) {
        success7++;
      } else if (desired == 8) {
        success8++;
      } else if (desired == 9) {
        success9++;
      }
    }
    std::cout << "Image:  " << i << "  Label:  " << desired
              << " Neural Net Result:  " << index
              << "  Neural Net Output:  " << outf << std::endl;
  }
  std::cout << "Distribution: " << std::endl
            << "0 " << success0 << std::endl
            << "1 " << success1 << std::endl
            << "2 " << success2 << std::endl
            << "3 " << success3 << std::endl
            << "4 " << success4 << std::endl
            << "5 " << success5 << std::endl
            << "6 " << success6 << std::endl
            << "7 " << success7 << std::endl
            << "8 " << success8 << std::endl
            << "9 " << success9 << std::endl;
  return success;
}

void GPUNetwork::train_network(float *data_set, size_t set_size,
                               float *data_labels, size_t label_size,
                               unsigned int dataset_count, unsigned int epochs,
                               const float learning_rate, float momentum) {

  checkErrorsCuda(
      hipMalloc((void **)&device_dataset, sizeof(float) * set_size));
  checkErrorsCuda(
      hipMalloc((void **)&device_labels, sizeof(float) * label_size));

  checkErrorsCuda(hipMemcpy(device_dataset, data_set, sizeof(float) * set_size,
                             hipMemcpyHostToDevice));
  checkErrorsCuda(hipMemcpy(device_labels, data_labels,
                             sizeof(float) * label_size,
                             hipMemcpyHostToDevice));
  hipProfilerStart();
  for (unsigned int e = 0; e < epochs; e++) {

    std::cout << "Epoch " << e + 1 << "/" << epochs << std::endl;

    for (unsigned int i = 0; i < dataset_count; i++) {

      device_awaited_output = device_labels + i * (label_size / dataset_count);

      Device::set_dataset<<<1, (set_size / dataset_count)>>>(
          device_inputs[0], device_dataset + (i * (set_size / dataset_count)),
          input_size[0]);

      // checkErrorsCuda(hipDeviceSynchronize());

      Device::tile_propagate_inlayer<<<num_blocks[0], threads_per_block[0]>>>(
          device_inputs[0], device_inputs[1], device_weights[0],
          device_wbias[0], input_size[0], neuron_size[0]);
      // checkErrorsCuda(hipDeviceSynchronize());

      for (unsigned int l = 1; l < count_layers; l++) {
        if (l >= count_layers - 1) {
           //std::cout << "l194 " << l << std::endl;
          Device::tile_propagate_layer<<<num_blocks[l], threads_per_block[l]>>>(
              device_inputs[l], device_output, device_weights[l],
              device_wbias[l], input_size[l], neuron_size[l]);
        } else {
          // std::cout << "l199 " << l << std::endl;
          Device::tile_propagate_layer<<<num_blocks[l], threads_per_block[l]>>>(
              device_inputs[l], device_inputs[l + 1], device_weights[l],
              device_wbias[l], input_size[l], neuron_size[l]);
        }
        // checkErrorsCuda(hipDeviceSynchronize());
      }

      Device::tile_outlayer_train<<<num_blocks[count_layers - 1],
                                    threads_per_block[count_layers - 1]>>>(
          device_output, device_delta[count_layers - 1],
          device_wbias[count_layers - 1], device_awaited_output, learning_rate,
          input_size[count_layers - 1], neuron_size[count_layers - 1]);
      // checkErrorsCuda(hipDeviceSynchronize());

      for (int l = (int)count_layers - 2; l > -1; l--) {
        // std::cout << "l215 " << l << std::endl;
        Device::tile_layer_delta<<<mul_num_blocks[l + 1],
                                   mul_threads_per_block[l + 1]>>>(
            device_delta_summands[l], device_weights[l + 1],
            device_delta[l + 1], input_size[l], neuron_size[l]);

        Device::reduction<<<mul_num_blocks[l], mul_threads_per_block[l]>>>(
            device_delta_summands[l], device_delta_summands_out[l],
            input_size[l] * neuron_size[l]);

        Device::tile_layer_train<<<num_blocks[l], threads_per_block[l]>>>(
            device_inputs[l + 1], device_delta_summands_out[l], device_wbias[l],
            device_delta[l], device_awaited_output, learning_rate,
            input_size[l + 1], neuron_size[l + 1], input_size[l],
            neuron_size[l]);
        // checkErrorsCuda(hipDeviceSynchronize());
      }

      for (unsigned int l = 0; l < count_layers; l++) {
        // std::cout << "l225 " << l << std::endl;
        Device::
            tile_update_layer<<<mul_num_blocks[l], mul_threads_per_block[l]>>>(
                device_inputs[l], device_weights[l], device_delta[l],
                device_prvdeltas[l], learning_rate, momentum, input_size[l],
                neuron_size[l]);
        // checkErrorsCuda(hipDeviceSynchronize());
      }
    }
  }
  hipProfilerStop();
}

float *GPUNetwork::getOutput() {

  float *iout = new float[10];
  checkErrorsCuda(hipMemcpy(iout, device_output, sizeof(float) * 10,
                             hipMemcpyDeviceToHost));
  for (int i = 0; i < 10; i++) {
    std::cout << "Output on Neuron " << i << " : " << iout[i] << std::endl;
  }
  return iout;
}

GPUNetwork::~GPUNetwork() {
  delete num_blocks;
  delete threads_per_block;
  delete neuron_size;
  delete input_size;

  // device_land
  for (unsigned int i = 0; i < count_layers; i++) {
    checkErrorsCuda(hipFree(device_inputs[i]));
    checkErrorsCuda(hipFree(device_weights[i]));
    checkErrorsCuda(hipFree(device_wbias[i]));
    checkErrorsCuda(hipFree(device_delta[i]));
    checkErrorsCuda(hipFree(device_delta_summands[i]));
    checkErrorsCuda(hipFree(device_prvdeltas[i]));
  }

  checkErrorsCuda(hipFree(device_output));
  checkErrorsCuda(hipFree(device_dataset));
  checkErrorsCuda(hipFree(test_device_dataset));
  checkErrorsCuda(hipFree(device_labels));

  delete device_inputs;
  delete device_weights;
  delete device_wbias;
  delete device_delta;
  delete device_delta_summands;
  delete device_prvdeltas;
}
