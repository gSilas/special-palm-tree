#include "hip/hip_runtime.h"
#include "gpunetwork.cuh"

__global__ void
Device::tile_update_layer(float *device_input, float *device_weights,
                          float *device_delta, float *device_prvdeltas,
                          float learning_rate, float momentum,
                          unsigned int input_offset, unsigned int neuron_offset,
                          unsigned int input_size, unsigned int weight_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;

  float dw;
  for (unsigned int j = 0; j < input_size; j++) {

    unsigned int index = j + weight_offset + tid_x * input_size;

    dw = learning_rate * device_input[j + input_offset] *
         device_delta[neuron_offset + tid_x];
    dw += momentum * device_prvdeltas[index];
    /*  printf("threadIdx: %d dw: %f input %f delta %f \n", tid_x, dw,
             device_input[j + input_offset], device_delta[neuron_offset +
       tid_x]); */

    device_prvdeltas[index] = dw;
    device_weights[index] += dw;
  }
}
__global__ void Device::tile_propagate_inlayer(
    float *data_set, float *device_input, float *device_weights,
    float *device_wbias, float *device_output, unsigned int input_size,
    unsigned int neuron_size, unsigned int nl_neuron_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;

  float output = 0;
  for (unsigned int j = 0; j < input_size; j++) {
    device_input[j] = data_set[j];
    output += device_weights[j + tid_x * input_size] *
              device_input[j];
    /*  printf(" prop threadIdx: %d we %f in %f input_off %d \n", tid_x,
             device_weights[j + weight_offset + tid_x * input_size],
             device_input[j + input_offset],input_offset); */
  }

  output += device_wbias[tid_x];
  /*printf("threadIdx: %d input_offset: %d neuron_offset: %d\n", tid_x,
         input_offset, neuron_offset);*/
  float res = 1 / (1 + exp(-output));
  device_output[tid_x] = res;
  device_input[nl_neuron_offset + tid_x] = res;
  // printf(" prop threadIdx: %d neuron_off %d \n", nl_neuron_offset);
}

__global__ void Device::tile_propagate_layer(
    float *device_input, float *device_weights, float *device_wbias,
    float *device_output, unsigned int input_size, unsigned int neuron_size,
    unsigned int input_offset, unsigned int neuron_offset,
    unsigned int nl_neuron_offset, unsigned int weight_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;
  unsigned int index = neuron_offset + tid_x;

  float output = 0;
  for (unsigned int j = 0; j < input_size; j++) {

    output += device_weights[j + weight_offset + tid_x * input_size] *
              device_input[j + input_offset];
    /*  printf(" prop threadIdx: %d we %f in %f input_off %d \n", tid_x,
             device_weights[j + weight_offset + tid_x * input_size],
             device_input[j + input_offset],input_offset); */
  }

  output += device_wbias[index];
  /*printf("threadIdx: %d input_offset: %d neuron_offset: %d\n", tid_x,
         input_offset, neuron_offset);*/
  float res = 1 / (1 + exp(-output));
  device_output[index] = res;
  device_input[nl_neuron_offset + tid_x] = res;
  // printf(" prop threadIdx: %d neuron_off %d \n", nl_neuron_offset);
}

__global__ void Device::tile_outlayer_train(float *device_delta,
                                            float *device_wbias,
                                            float *device_output,
                                            float *device_awaited_output,
                                            float learning_rate,
                                            unsigned int nl_neuron_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;
  unsigned int index = nl_neuron_offset + tid_x;
  float out;
  // printf("threadIdx: %d neuron_offset: %d\n", tid_x, nl_neuron_offset);
  out = device_output[index];
  float delta = (device_awaited_output[tid_x] - out) * out * (1 - out);
  device_delta[index] = delta;

  device_wbias[index] += learning_rate * delta;
}

__global__ void Device::tile_layer_train(
    float *device_weights, float *device_wbias, float *device_delta,
    float *device_output, float *device_awaited_output, float learning_rate,
    unsigned int pl_neuron_size, unsigned int pl_input_size,
    unsigned int tl_weight_offset, unsigned int tl_neuron_offset,
    unsigned int nl_neuron_offset) {

  int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  // int tid_y = blockIdx.y*blockDim.y+threadIdx.y;
  unsigned int index = nl_neuron_offset + tid_x;

  float out;
  float delta = 0;
  // nl = layer[this-1]| tl = layer[this] | pl = layer[this + 1]
  for (unsigned int i = 0; i < pl_neuron_size; i++)
    for (unsigned int j = 0; j < pl_input_size; j++) {
      delta += device_weights[tl_weight_offset + i * pl_input_size + j] *
               device_delta[tl_neuron_offset + i];
    }

  out = device_output[index];
  // printf("threadIdx: %d neuron_offset: %d\n", tid_x, nl_neuron_offset);
  float rdelta = out * (1 - out) * delta;
  device_delta[index] = rdelta;
  device_wbias[index] += learning_rate * rdelta;
}

void GPUNetwork::init_network(unsigned int *inputs, unsigned int *neurons,
                              unsigned int clayers) {

  count_layers = clayers;
  arr_input_size = new unsigned int[clayers];
  arr_neuron_size = new unsigned int[clayers];
  sum_input_size = new unsigned int[clayers + 1];
  sum_neuron_size = new unsigned int[clayers + 1];
  sum_weight_size = new unsigned int[clayers + 1];
  sum_input_size[0] = 0;
  sum_neuron_size[0] = 0;
  sum_weight_size[0] = 0;
  num_blocks = new int[clayers];
  threads_per_block = new int[clayers];

  std::memcpy(arr_input_size, inputs, sizeof(unsigned int) * clayers);
  std::memcpy(arr_neuron_size, neurons, sizeof(unsigned int) * clayers);

  hipDeviceProp_t device_props;
  checkErrorsCuda(hipGetDeviceProperties(&device_props, 0));

  size_t input_size = 0;
  size_t neuron_size = 0;
  size_t weight_size = 0;

  for (unsigned int l = 0; l < clayers; l++) {
    input_size += inputs[l];
    neuron_size += neurons[l];

    weight_size += inputs[l] * neurons[l];
    if (neurons[l] > (unsigned int)device_props.maxThreadsPerBlock)
      threads_per_block[l] = device_props.maxThreadsPerBlock;
    else
      threads_per_block[l] = neurons[l];

    num_blocks[l] = (int)(neurons[l] / threads_per_block[l]);

    if (num_blocks[l] == 0)
      num_blocks[l]++;
    else if (0 != neurons[l] % num_blocks[l])
      num_blocks[l]++;

    std::cout << "num_blocks = " << num_blocks[l] << std::endl;
  }

  for (unsigned int l = 1; l < clayers + 1; l++) {
    sum_input_size[l] = inputs[l - 1] + sum_input_size[l - 1];
    sum_neuron_size[l] = neurons[l - 1] + sum_neuron_size[l - 1];
    sum_weight_size[l] = neurons[l - 1] * inputs[l - 1];
  }

  for (unsigned int l = 0; l < clayers + 1; l++) {
    std::cout << l << " | " << sum_input_size[l] << " | " << sum_neuron_size[l]
              << std::endl;
  }

  std::cout << input_size << " | " << neuron_size << " | " << weight_size
            << std::endl
            << "hipMalloc" << std::endl;

  checkErrorsCuda(hipMalloc(&device_input, sizeof(float) * input_size));

  checkErrorsCuda(hipMalloc(&device_wbias, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMalloc(&device_output, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMalloc(&device_delta, sizeof(float) * neuron_size));

  checkErrorsCuda(hipMalloc(&device_weights, sizeof(float) * weight_size));
  checkErrorsCuda(hipMalloc(&device_prvdeltas, sizeof(float) * weight_size));

  std::cout << "hipMemset" << std::endl;

  checkErrorsCuda(hipMemset(device_input, 0, sizeof(float) * input_size));

  checkErrorsCuda(hipMemset(device_wbias, 0, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMemset(device_output, 0, sizeof(float) * neuron_size));
  checkErrorsCuda(hipMemset(device_delta, 0, sizeof(float) * neuron_size));

  checkErrorsCuda(hipMemset(device_weights, 1, sizeof(float) * weight_size));
  checkErrorsCuda(hipMemset(device_prvdeltas, 0, sizeof(float) * weight_size));
}
unsigned int GPUNetwork::propagate_network(float *data_set, float *label_set,
                                           unsigned int dataset_count,
                                           size_t set_size, size_t label_size) {
  unsigned int success = 0;

  checkErrorsCuda(hipMalloc(&test_device_dataset, sizeof(float) * set_size));

  checkErrorsCuda(hipMemcpy(test_device_dataset, data_set,
                             sizeof(float) * set_size, hipMemcpyHostToDevice));

  for (unsigned int i = 0; i < dataset_count; i++) {
    Device::tile_propagate_inlayer<<<num_blocks[0], threads_per_block[0]>>>(
        test_device_dataset + i * (set_size / dataset_count), device_input,
        device_weights, device_wbias, device_output, arr_input_size[0],
        arr_neuron_size[0], sum_neuron_size[1]);

    // checkErrorsCuda(hipDeviceSynchronize());

    for (int l = 2; l < count_layers + 1; l++) {

      Device::
          tile_propagate_layer<<<num_blocks[l - 1], threads_per_block[l - 1]>>>(
              device_input, device_weights, device_wbias, device_output,
              arr_input_size[l - 1], arr_neuron_size[l - 1],
              sum_input_size[l - 1], sum_neuron_size[l - 1], sum_neuron_size[l],
              sum_weight_size[l - 1]);

      /*  std::cout << l << " | " << num_blocks[l - 1] << " | "
                  << threads_per_block[l - 1] << " | " << arr_input_size[l - 1]
                  << " | " << arr_neuron_size[l - 1] << " | "
                  << sum_input_size[l - 1] << " | " << sum_neuron_size[l - 1]
                  << " | " << sum_neuron_size[l] << " | "
                  << sum_weight_size[l - 1] << std::endl;*/

      // checkErrorsCuda(hipDeviceSynchronize());
    }
    float *out;
    out = getOutput();

    float outf = 0;

    for (int j = 0; j < 10; j++) {
      outf += out[j + sum_neuron_size[count_layers - 1]];
    }
    float desired = 0;
    for (int j = 0; j < 10; j++) {
      desired += label_set[j + i * (label_size / dataset_count)];
    }

    if (std::round(outf) == desired)
      success++;
    std::cout << "TESTED PATTERN " << i << " DESIRED OUTPUT: " << desired
              << " NET RESULT: " << outf << std::endl;
  }
  return success;
}

void GPUNetwork::train_network(float *data_set, size_t set_size,
                               float *data_labels, size_t label_size,
                               unsigned int dataset_count, unsigned int epochs,
                               const float learning_rate, float momentum) {

  checkErrorsCuda(hipMalloc(&device_dataset, sizeof(float) * set_size));
  checkErrorsCuda(hipMalloc(&device_labels, sizeof(float) * label_size));

  checkErrorsCuda(hipMemcpy(device_dataset, data_set, sizeof(float) * set_size,
                             hipMemcpyHostToDevice));
  checkErrorsCuda(hipMemcpy(device_labels, data_labels,
                             sizeof(float) * label_size,
                             hipMemcpyHostToDevice));
  for (unsigned int e = 0; e < epochs; e++) {
    std::cout << "EPOCH " << e << std::endl;
    for (unsigned int i = 0; i < dataset_count; i++) {

      device_awaited_output = device_labels + i * (label_size / dataset_count);

      Device::tile_propagate_inlayer<<<num_blocks[0], threads_per_block[0]>>>(
          device_dataset + i * (set_size / dataset_count), device_input,
          device_weights, device_wbias, device_output, arr_input_size[0],
          arr_neuron_size[0], sum_neuron_size[1]);

      // checkErrorsCuda(hipDeviceSynchronize());

      for (int l = 1; l < count_layers - 1; l++) {

        Device::tile_propagate_layer<<<num_blocks[l - 1],
                                       threads_per_block[l - 1]>>>(
            device_input, device_weights, device_wbias, device_output,
            arr_input_size[l - 1], arr_neuron_size[l - 1],
            sum_input_size[l - 1], sum_neuron_size[l - 1], sum_neuron_size[l],
            sum_weight_size[l - 1]);

        // checkErrorsCuda(hipDeviceSynchronize());
        /*std::cout << l << " | " << num_blocks[l] << " | " <<
           threads_per_block[l]
                  << " | " << arr_input_size[l] << " | " << arr_neuron_size[l]
                  << " | " << sum_input_size[l] << " | " << sum_neuron_size[l]
                  << " | " << sum_weight_size[l] << " | " << std::endl;*/
      }

      Device::tile_outlayer_train<<<num_blocks[count_layers - 1],
                                    threads_per_block[count_layers - 1]>>>(
          device_delta, device_wbias, device_output, device_awaited_output,
          learning_rate, sum_neuron_size[count_layers - 1]);

      // checkErrorsCuda(hipDeviceSynchronize());

      for (int l = count_layers - 2; l >= 0; l--) {
        Device::tile_layer_train<<<num_blocks[l], threads_per_block[l]>>>(
            device_weights, device_wbias, device_delta, device_output,
            device_awaited_output, learning_rate, arr_neuron_size[l + 1],
            arr_input_size[l + 1], sum_weight_size[l + 1],
            sum_neuron_size[l + 1], sum_neuron_size[l]);

        // checkErrorsCuda(hipDeviceSynchronize());
        /*
            std::cout << " tile_layer_train " << std::endl;
            std::cout << l << " | " << num_blocks[l] << " | " <<
           threads_per_block[l]
                      << " | " << arr_neuron_size[l + 1] << " | "
                      << arr_input_size[l + 1] << " | " << sum_weight_size[l +
           1]
                      << " | " << sum_neuron_size[l + 1] << " | " <<
           sum_neuron_size[l]
                      << std::endl; */
      }

      for (int l = 0; l < count_layers; l++) {
        Device::tile_update_layer<<<num_blocks[l], threads_per_block[l]>>>(
            device_input, device_weights, device_delta, device_prvdeltas,
            learning_rate, momentum, sum_input_size[l], sum_neuron_size[l],
            arr_input_size[l], sum_weight_size[l]);

        // checkErrorsCuda(hipDeviceSynchronize());
        /*
            std::cout << " tile_update_layer " << std::endl;
            std::cout << l << " | " << num_blocks[l] << " | " <<
           threads_per_block[l]
                      << " | " << sum_input_size[l] << " | " <<
           sum_neuron_size[l]
                      << " | " << arr_input_size[l] << " | " <<
           sum_weight_size[l]
                      << std::endl; */
      }
    }
  }
}

float *GPUNetwork::getOutput() {
  float *out = new float[sum_neuron_size[count_layers]];

  checkErrorsCuda(hipMemcpy(out, device_output,
                             sum_neuron_size[count_layers] * sizeof(float),
                             hipMemcpyDeviceToHost));

  return out;
}

GPUNetwork::~GPUNetwork() {
  delete num_blocks;
  delete threads_per_block;
  delete arr_input_size;
  delete arr_neuron_size;
  delete sum_input_size;
  delete sum_neuron_size;
  delete sum_weight_size;

  // device_land
  checkErrorsCuda(hipFree(device_input));

  checkErrorsCuda(hipFree(device_weights));
  checkErrorsCuda(hipFree(device_wbias));

  checkErrorsCuda(hipFree(device_delta));
  checkErrorsCuda(hipFree(device_prvdeltas));
  checkErrorsCuda(hipFree(device_output));

  checkErrorsCuda(hipFree(device_dataset));
  checkErrorsCuda(hipFree(device_labels));

  checkErrorsCuda(hipFree(test_device_dataset));
}
