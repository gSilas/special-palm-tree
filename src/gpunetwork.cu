#include "gpunetwork.cuh"

void GPUNetwork::init_network(unsigned int *inputs, unsigned int *neurons,
                              unsigned int clayers) {

  count_layers = clayers;
  num_blocks = new int[clayers];
  threads_per_block = new int[clayers];

  hipDeviceProp_t device_props;
  checkErrorsCuda(hipGetDeviceProperties(&device_props, 0));
  for (unsigned int l = 0; l < clayers; l++) {
    if (neurons[l] > (unsigned int)device_props.maxThreadsPerBlock)
      threads_per_block[l] = device_props.maxThreadsPerBlock;
    else
      threads_per_block[l] = neurons[l];

    num_blocks[l] = (int)(neurons[l] / threads_per_block[l]);

    if (num_blocks[l] == 0)
      num_blocks[l]++;
    else if (0 != neurons[l] % num_blocks[l])
      num_blocks[l]++;

    //std::cout << "num_blocks = " << num_blocks[l] << std::endl;
  }
  device_inputs = new float *[clayers];

  device_weights = new float *[clayers];
  device_wbias = new float *[clayers];

  device_delta = new float *[clayers];
  device_prvdeltas = new float *[clayers];

  checkErrorsCuda(hipMalloc((void **)&device_output,
                             sizeof(float) * neurons[count_layers - 1]));

  for (int i = 0; i < clayers; i++) {
    checkErrorsCuda(
        hipMalloc((void **)&device_inputs[i], sizeof(float) * inputs[i]));
    checkErrorsCuda(
        hipMalloc((void **)&device_wbias[i], sizeof(float) * neurons[i]));
    checkErrorsCuda(
        hipMalloc((void **)&device_delta[i], sizeof(float) * neurons[i]));

    checkErrorsCuda(hipMalloc((void **)&device_weights[i],
                               sizeof(float) * inputs[i] * neurons[i]));
    checkErrorsCuda(hipMalloc((void **)&device_prvdeltas[i],
                               sizeof(float) * inputs[i] * neurons[i]));

    Device::set_layer_memory<<<num_blocks[i], threads_per_block[i]>>>(
        device_delta[i], device_prvdeltas[i], inputs[i], neurons[i]);

    float *weights = new float[inputs[i] * neurons[i]];
    float *wbias = new float[neurons[i]];

    for (unsigned int j = 0; j < inputs[i] * neurons[i]; j++) {
      weights[j] = -0.5 +
                   static_cast<float>(rand()) /
                       (static_cast<float>(RAND_MAX / (0.5 - (-0.5))));
    }
    for (unsigned int j = 0; j < neurons[i]; j++) {
      wbias[j] = -0.5 +
                 static_cast<float>(rand()) /
                     (static_cast<float>(RAND_MAX / (0.5 - (-0.5))));
    }

    checkErrorsCuda(hipMemcpy(device_weights[i], weights,
                               sizeof(float) * inputs[i] * neurons[i],
                               hipMemcpyHostToDevice));
    checkErrorsCuda(hipMemcpy(device_wbias[i], wbias,
                               sizeof(float) * neurons[i],
                               hipMemcpyHostToDevice));
    delete weights;
    delete wbias;
  }
}

unsigned int GPUNetwork::propagate_network(float *data_set, float *label_set,
                                           unsigned int dataset_count,
                                           size_t set_size, size_t label_size) {
  unsigned int success = 0;

  checkErrorsCuda(
      hipMalloc((void **)&test_device_dataset, sizeof(float) * set_size));

  checkErrorsCuda(hipMemcpy(test_device_dataset, data_set,
                             sizeof(float) * set_size, hipMemcpyHostToDevice));

  for (unsigned int i = 0; i < dataset_count; i++) {

    Device::tile_propagate_inlayer<<<num_blocks[0], threads_per_block[0]>>>(
        test_device_dataset + (i * (set_size / dataset_count)),
        device_inputs[0], device_inputs[1], device_weights[0], device_wbias[0],
        784, 300);
    checkErrorsCuda(hipDeviceSynchronize());
    Device::tile_propagate_layer<<<num_blocks[1], threads_per_block[1]>>>(
        device_inputs[1], device_output, device_weights[1], device_wbias[1],
        300, 10);
    checkErrorsCuda(hipDeviceSynchronize());

    float *out;
    out = getOutput();

    float outf = -1;
    float index = 0;

    for (int j = 0; j < 10; j++) {
      if (out[j] > outf) {
        outf = out[j];
        index = j;
      }
    }

    float desired = 0;
    for (int j = 0; j < 10; j++) {
      if (label_set[j + i * (label_size / dataset_count)] == 1) {
        desired = j;
      }
    }

    if (index == desired)
      success++;
    std::cout << "Pattern:  " << i << "  Label:  " << desired
              << " Neural Net Result:  " << index
              << "  Neural Net Output:  " << outf << std::endl;
  }
  return success;
}

void GPUNetwork::train_network(float *data_set, size_t set_size,
                               float *data_labels, size_t label_size,
                               unsigned int dataset_count, unsigned int epochs,
                               const float learning_rate, float momentum) {

  checkErrorsCuda(
      hipMalloc((void **)&device_dataset, sizeof(float) * set_size));
  checkErrorsCuda(
      hipMalloc((void **)&device_labels, sizeof(float) * label_size));

  checkErrorsCuda(hipMemcpy(device_dataset, data_set, sizeof(float) * set_size,
                             hipMemcpyHostToDevice));
  checkErrorsCuda(hipMemcpy(device_labels, data_labels,
                             sizeof(float) * label_size,
                             hipMemcpyHostToDevice));
  for (unsigned int e = 0; e < epochs; e++) {

    std::cout << "Epoch " << e + 1 << "/" << epochs << std::endl;
    for (unsigned int i = 0; i < dataset_count; i++) {

      device_awaited_output = device_labels + i * (label_size / dataset_count);

      Device::tile_propagate_inlayer<<<num_blocks[0], threads_per_block[0]>>>(
          device_dataset + (i * (set_size / dataset_count)), device_inputs[0],
          device_inputs[1], device_weights[0], device_wbias[0], 784, 300);
      checkErrorsCuda(hipDeviceSynchronize());
      Device::tile_propagate_layer<<<num_blocks[1], threads_per_block[1]>>>(
          device_inputs[1], device_output, device_weights[1], device_wbias[1],
          300, 10);
      checkErrorsCuda(hipDeviceSynchronize());

      Device::tile_outlayer_train<<<num_blocks[1], threads_per_block[1]>>>(
          device_output, device_delta[1], device_wbias[1],
          device_awaited_output, learning_rate, 300, 10);
      checkErrorsCuda(hipDeviceSynchronize());
      Device::tile_layer_train<<<num_blocks[0], threads_per_block[0]>>>(
          device_inputs[1], device_weights[1], device_delta[1], device_wbias[0],
          device_delta[0], device_awaited_output, learning_rate, 300, 10, 784,
          300);
      checkErrorsCuda(hipDeviceSynchronize());

      Device::tile_update_layer<<<num_blocks[0], threads_per_block[0]>>>(
          device_inputs[0], device_weights[0], device_delta[0],
          device_prvdeltas[0], learning_rate, momentum, 784, 300);
      checkErrorsCuda(hipDeviceSynchronize());
      Device::tile_update_layer<<<num_blocks[1], threads_per_block[1]>>>(
          device_inputs[1], device_weights[1], device_delta[1],
          device_prvdeltas[1], learning_rate, momentum, 300, 10);
      checkErrorsCuda(hipDeviceSynchronize());
    }
  }
}

float *GPUNetwork::getOutput() {
  // float *out = new float[sum_neuron_size[count_layers]];
  /*float *iout = new float[sum_input_size[count_layers] +
                          arr_neuron_size[count_layers - 1]];
  float *wout = new float[sum_weight_size[count_layers]];
  float *dout = new float[sum_neuron_size[count_layers]];

  checkErrorsCuda(hipMemcpy(dout, device_delta,
                             sizeof(float) * sum_neuron_size[count_layers],
                             hipMemcpyDeviceToHost));
  checkErrorsCuda(hipMemcpy(iout, device_input,
                             sizeof(float) * sum_input_size[count_layers] +
                                 arr_neuron_size[count_layers - 1],
                             hipMemcpyDeviceToHost));
  checkErrorsCuda(hipMemcpy(wout, device_weights,
                             sizeof(float) * sum_weight_size[count_layers],
                             hipMemcpyDeviceToHost));
  for (int i = 0; i < sum_neuron_size[count_layers]; i++) {
    std::cout << "d " << i << " " << dout[i] << std::endl;
  }

  for (int i = 0;
       i < sum_input_size[count_layers] + arr_neuron_size[count_layers - 1];
       i++) {
    std::cout << "i " << i << " " << iout[i] << std::endl;
  }
  for (int i = 0; i < sum_weight_size[count_layers]; i++) {
    std::cout << "w " << i << " " << wout[i] << std::endl;
  }
  delete iout;
  delete wout;
  delete dout;*/
  float *iout = new float[10];
  checkErrorsCuda(hipMemcpy(iout, device_output, sizeof(float) * 10,
                             hipMemcpyDeviceToHost));
  for (int i = 0; i < 10; i++) {
    std::cout << "Output on Neuron " << i << " : " << iout[i] << std::endl;
  }
  return iout;
}

GPUNetwork::~GPUNetwork() {
  /*  delete num_blocks;hipMemsetD32
    delete threads_per_block;
    delete arr_input_size;
    delete arr_neuron_size;
    delete sum_input_size;
    delete sum_neuron_size;
    delete sum_weight_size;

    // device_land
    checkErrorsCuda(hipFree(device_input));

    checkErrorsCuda(hipFree(device_weights));
    checkErrorsCuda(hipFree(device_wbias));

    checkErrorsCuda(hipFree(device_delta));
    checkErrorsCuda(hipFree(device_prvdeltas));

    checkErrorsCuda(hipFree(device_dataset));
    checkErrorsCuda(hipFree(device_labels));

    checkErrorsCuda(hipFree(test_device_dataset)); */
}
